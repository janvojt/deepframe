/* 
 * File:   MseErrorComputer.cpp
 * Author: janvojt
 * 
 * Created on November 29, 2014, 12:58 PM
 */

#include "cudaDebugHelpers.h"

#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


void dumpDeviceArray(char flag, data_t *dm, int size) {
    std::cout << flag << std::endl;
    data_t *hdm = new data_t[size];
    checkCudaErrors(hipMemcpy(hdm, dm, sizeof(data_t) * size, hipMemcpyDeviceToHost));
    
    for (int i = 0; i<size; i++) {
        std::cout << "Dumping device " << flag << ": " << hdm[i] << std::endl;
    }
    std::cout << "-----------------------------" << std::endl;
    
    delete[] hdm;
}


void compare(char flag, double *dm, double *hm, int size) {
    double *hdm = new double[size];
    checkCudaErrors(hipMemcpy(hdm, dm, sizeof(double) * size, hipMemcpyDeviceToHost));
    
    for (int i = 0; i<size; i++) {
        if (hdm[i] == hm[i]) {
            std::cout << "Comparing " << flag << ": " << hdm[i] << " =?= " << hm[i] << std::endl;
        } else {
            std::cout << "Comparing " << flag << ": " << hdm[i] << " =?= " << hm[i] << "        !!!!!!!!!!!!!!!!!!" << std::endl;
        }
    }
    
    delete[] hdm;
}
