/* 
 * File:   MseErrorComputer.cpp
 * Author: janvojt
 * 
 * Created on November 29, 2014, 12:58 PM
 */

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>


__global__
void sumVectors(double *dA, double *dB, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < elements) {
        dA[i] += dB[i];
    }
}
void k_sumVectors(double *dA, double *dB, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    sumVectors<<<bs,ts>>>(dA, dB, elements);
}

__global__
void computeOutputLocalGradient(double *actualOutput, double *expectedOutput, double *localGradient, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elements) {
        double derivative = actualOutput[i] * (1.0 - actualOutput[i]);
        localGradient[i] = (actualOutput[i] - expectedOutput[i]) * derivative;
    }
}
void k_computeOutputLocalGradient(double *actualOutput, double *expectedOutput, double *localGradient, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    computeOutputLocalGradient<<<bs,ts>>>(actualOutput, expectedOutput, localGradient, elements);
}

__global__
void computeTotalDerivative(double learningRate, int nextNeurons,
        double *thisInput, double *nextLocalGradient,
        double *weightDiffs, int elements) {
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < elements) {
        int i = idx / nextNeurons;
        int j = idx % nextNeurons;
        weightDiffs[i*nextNeurons+j] = -learningRate * nextLocalGradient[j] * thisInput[i];
    }
}
void k_computeTotalDerivative(int thisNeurons, int nextNeurons, 
        double learningRate, double *thisInput, double *nextLocalGradient,
        double *weightDiffs) {
    int ts = 512;
    int bs = (thisNeurons * nextNeurons + ts - 1) / ts;
    computeTotalDerivative<<<bs,ts>>>(learningRate, nextNeurons,
        thisInput, nextLocalGradient,
        weightDiffs, thisNeurons * nextNeurons);
}

__global__
void computeBiasDerivative(double learningRate, double *nextLocalGradient,
        double *biasDiffs) {
    
    int i = threadIdx.x;
    
    biasDiffs[i] = -learningRate * nextLocalGradient[i];
}
void k_computeBiasDerivative(const dim3 bs, const dim3 ts, 
        double learningRate, double *nextLocalGradient,
        double *biasDiffs) {
    computeBiasDerivative<<<bs,ts>>>(learningRate, nextLocalGradient,
        biasDiffs);
}

__global__
void computeHiddenLocalGradient(int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    int i = threadIdx.x;
    
    double derivative = thisInput[i] * (1.0 - thisInput[i]);
    
    double sumNextGradient = 0;
    for (int j = 0; j<nextNeurons; j++) {
        sumNextGradient += nextLocalGradient[j] * weights[i * nextNeurons + j];
    }
    thisLocalGradient[i] = sumNextGradient * derivative;
}
void k_computeHiddenLocalGradient(const dim3 bs, const dim3 ts, int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    computeHiddenLocalGradient<<<bs,ts>>>(nextNeurons,
        thisInput, weights,
        thisLocalGradient, nextLocalGradient);
}


__global__
void clearLayer(double *valuePtr) {
    valuePtr[threadIdx.x] = 0;
}
void k_clearLayer(const dim3 bs, const dim3 ts, double *valuePtr) {
    clearLayer<<<bs,ts>>>(valuePtr);
}


__global__
void sumArrays(double *dA, double *dB) {
    dA[threadIdx.x] += dB[threadIdx.x];
}
void k_sumArrays(const dim3 bs, const dim3 ts, double *dA, double *dB) {
    sumArrays<<<bs,ts>>>(dA, dB);
}


__global__
void computeSigmoid(double *dArray) {
	int i = threadIdx.x;
	dArray[i] = 1.0 / (1.0 + exp(-dArray[i]));
}
void k_computeSigmoid(const dim3 bs, const dim3 ts, double *dArray) {
	computeSigmoid<<<bs,ts>>>(dArray);
}