/* 
 * File:   MseErrorComputer.cpp
 * Author: janvojt
 * 
 * Created on November 29, 2014, 12:58 PM
 */

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>


__global__
void sumVectors(double *dA, double *dB, int elements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < elements) {
        dA[i] += dB[i];
    }
}
void k_sumVectors(double *dA, double *dB, int elements) {
    int ts = 512;
    int bs = (elements + ts - 1) / ts;
    sumVectors<<<bs,ts>>>(dA, dB, elements);
}

__global__
void computeOutputLocalGradient(double *actualOutput, double *expectedOutput, double *localGradient) {
    int i = threadIdx.x;
    double derivative = actualOutput[i] * (1.0 - actualOutput[i]);
    localGradient[i] = (actualOutput[i] - expectedOutput[i]) * derivative;
}
void k_computeOutputLocalGradient(const dim3 bs, const dim3 ts, double *actualOutput, double *expectedOutput, double *localGradient) {
    computeOutputLocalGradient<<<bs,ts>>>(actualOutput, expectedOutput, localGradient);
}

__global__
void computeTotalDerivative(double learningRate, int nextNeurons,
        double *thisInput, double *nextLocalGradient,
        double *weightDiffs) {
    
    int i = threadIdx.x;
    int j = threadIdx.y;

    weightDiffs[i*nextNeurons+j] = -learningRate * nextLocalGradient[j] * thisInput[i];
}
void k_computeTotalDerivative(const dim3 bs, const dim3 ts, 
        double learningRate, int nextNeurons,
        double *thisInput, double *nextLocalGradient,
        double *weightDiffs) {
    computeTotalDerivative<<<bs,ts>>>(learningRate, nextNeurons,
        thisInput, nextLocalGradient,
        weightDiffs);
}

__global__
void computeBiasDerivative(double learningRate, double *nextLocalGradient,
        double *biasDiffs) {
    
    int i = threadIdx.x;
    
    biasDiffs[i] = -learningRate * nextLocalGradient[i];
}
void k_computeBiasDerivative(const dim3 bs, const dim3 ts, 
        double learningRate, double *nextLocalGradient,
        double *biasDiffs) {
    computeBiasDerivative<<<bs,ts>>>(learningRate, nextLocalGradient,
        biasDiffs);
}

__global__
void computeHiddenLocalGradient(int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    int i = threadIdx.x;
    
    double derivative = thisInput[i] * (1.0 - thisInput[i]);
    
    double sumNextGradient = 0;
    for (int j = 0; j<nextNeurons; j++) {
        sumNextGradient += nextLocalGradient[j] * weights[i * nextNeurons + j];
    }
    thisLocalGradient[i] = sumNextGradient * derivative;
}
void k_computeHiddenLocalGradient(const dim3 bs, const dim3 ts, int nextNeurons,
        double *thisInput, double *weights,
        double *thisLocalGradient, double *nextLocalGradient) {
    
    computeHiddenLocalGradient<<<bs,ts>>>(nextNeurons,
        thisInput, weights,
        thisLocalGradient, nextLocalGradient);
}


__global__
void clearLayer(double *valuePtr) {
    valuePtr[threadIdx.x] = 0;
}
void k_clearLayer(const dim3 bs, const dim3 ts, double *valuePtr) {
    clearLayer<<<bs,ts>>>(valuePtr);
}


__global__
void sumArrays(double *dA, double *dB) {
    dA[threadIdx.x] += dB[threadIdx.x];
}
void k_sumArrays(const dim3 bs, const dim3 ts, double *dA, double *dB) {
    sumArrays<<<bs,ts>>>(dA, dB);
}


__global__
void computeSigmoid(double *dArray) {
	int i = threadIdx.x;
	dArray[i] = 1.0 / (1.0 + exp(-dArray[i]));
}
void k_computeSigmoid(const dim3 bs, const dim3 ts, double *dArray) {
	computeSigmoid<<<bs,ts>>>(dArray);
}